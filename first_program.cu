__global__
void saveIDs(int *idsOut){
	const int tid = threadIdx.x;
	
	idsOut[tid] = tid;
}


#include <hip/hip_runtime.h>
#include <cstdlib>
#include <iostream>

int main(int argc, char **argv){
	const int numThreads = atoi(argv[1]);
	
	int *dIDs;
	
	hipMalloc(&dIDs, sizeof(int) * numThreads);
	
	saveIDs <<< 1, numThreads >>>(dIDs);
	
	int *hIDs = new int [numThreads];

	hipMemcpy (hIDs, dIDs, sizeof(int) * numThreads, hipMemcpyDeviceToHost);
	
	for (int i = 0; i < numThreads; ++i){
		std::cout << i << ": " << hIDs[i] << std::endl;
	}
	
	delete[] hIDs;
	hipFree(dIDs);
	
	return 0;
}

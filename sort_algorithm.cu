#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <iostream>

void checkCUDAError(const char *msg);

__global__ void product_two_vectors(int *d_a, int *d_b, int *d_r)
{
	int actual = blockIdx.x + threadIdx.x;
	d_r[actual] = d_a[actual] * d_b[actual];
}


int main(){

	int size_a = 100 * 1024;
	int *h_a;
	int *d_a, *d_b, *d_r;

	int numThreadsPerBlock = 100;
	int numBlocks = size_a / numThreadsPerBlock;

	malloc(size_a * sizeof(int));

	for (int i = 0; i < size_a; ++i) h_a[i] = i;
	hipMalloc(&d_a, size_a * sizeof(int));
	hipMalloc(&d_b, size_a * sizeof(int));
	hipMalloc(&d_r, size_a * sizeof(int));
	checkCUDAError("Error creating space");

	hipMemcpy(d_a, h_a, size_a * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("memcpy");
	for (int i = 0; i < size_a; ++i) h_a[i] = size_a - i - 1;

	hipMemcpy(d_b, h_a, size_a * sizeof(int), hipMemcpyHostToDevice);
	checkCUDAError("memcpy");

	dim3 blocks(numBlocks);
	dim3 threads(numThreadsPerBlock);

	sort_Array <<< blocks, threads >>> (d_a, d_b, d_r);
	cudaThreadSyncronize();

	cudamMemcpy(h_a, d_r, size_a * sizeof(int), hipMemcpyDeviceToHost);
	checkCUDAError("memcpy");

	for (int i = 0; i < count; ++i) 
	{
		assert (h_a[i] = i * (size_a - i));
		cout << h_a[i] << ",";
	}
	hipFree(d_a);
	hipFree(d_b);
	checkCUDAError("free_mem");
	free(h_a);

}

void checkCUDAError(const char *msg){

	hipError_t err = hipGetLastError();
	if(hipSuccess != err){
		fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) );
        exit(EXIT_FAILURE);
	}

}

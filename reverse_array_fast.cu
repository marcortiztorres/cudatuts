#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <string>

using namespace std;


void checkCudaError(string &s){
	hipError_t err = hipGetLastError();
	if(hipSuccess != err){
		cout << "Error" << s << endl;
	}
}
 
 __global__ void reverseArray(int *d_in, int *d_out)
 {

 	extern __shared__ int s_data[];

 	int posIn = blockDim * blockIdx.x + threadIdx.x;
 	s_data[blockDim - 1 - threadIdx.x] = d_in[posIn];

 	__syncthreads(); //Thanks GOD

 	int posOut = blockDim.x * (gridDim.x - 1 - blockIdx.x) + threadIdx.x;
    	d_out[posOut] = s_data[threadIdx.x];

 }



 int main(){

 	int *h_a, *d_a, *d_b;
 	int num_elem = 256 * 1000;

 	h_a = malloc(num_elem * sizeof(int));

 	for (int i = 0; i < num_elem; ++i)
 	{
 		h_a[i] = i;
 	}

 	int numThreadsPerBlock = 256;
 	int numBlocks = num_elem / numThreadsPerBlock;
 	int sharedMemSize = numThreadsPerBlock * sizeof(int);

 	hipMalloc(&d_a, num_elem * sizeof(int));
 	hipMalloc(&d_b, num_elem * sizeof(int));
 	checkCudaError("malloc");

 	hipMemcpy(d_a, h_a, num_elem * sizeof(int), hipMemcpyHostToDevice);
 	checkCudaError("memcpy");

 	dim3 blockDim(numThreadsPerBlock);
 	dim3 gridDim(numBlocks);
 	reverseArray <<< gridDim, blockDim, sharedMemSize >>> (d_a, d_b);

 	cudaThreadSyncronize();

 	hipMemcpy(h_a, d_b, num_elem * sizeof(int), hipMemcpyDeviceToHost);
 	checkCudaError("mempcy"); 

 	for (int i = 0; i < num_elem; ++i) 
 	{
 		assert (h_a[i] == num_elem - 1 - i);
 		cout << 
 	}

 	hipFree(d_a);
 	hipFree(d_b);
 	free(h_a);

 }

#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

void CheckCudaError(string &e);

__global__ void productMatrix(int *matrix_a, int *matrix_b, int *matrix_c)
{
	int suma;

	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	for (int i = 0; i < 100; ++i)
	{
		suma += matrix_a[row * 100 + i] * matrix_b[col * 100 + i];
	}
	matrix_c[(blockIdx.x * blockDim.x + threadIdx) * (blockIdx.y * blockDim.y + threadIdx.y)];
}

#define BLOCK_SIZE 5

int main(){

	//Creamos punteros para apuntar tanto al dispositivo como a memoria.
	int *h_a, *h_b;
	int *d_a, *d_b, *d_c;

	/*Trabajaremos con 20 blocks 
		es preferible blocks >= threads
		que blocks < threads
	*/

	int NumBlocks = 100 / BLOCK_SIZE;
	int num_elements = NumBlocks * BLOCK_SIZE;


	//Apuntamos los punteros hacia un espacio de 100*100 elementos en el host
	h_a = malloc(num_elements * sizeof(int));
	h_b = malloc(num_elements * sizeof(int));
	CheckCudaError("malloc_host_error");


	//LLenamos la memoria
	for (int i = 0; i < num_elements; ++i)
	{
		h_a[i] = i;
		h_b[i] = num_elements - 1 - i;
	}


	//Apuntamos los punteros del dispositivo hacia una reserva de memoria de 100*100 elementos.
	hipMalloc(&d_a, num_elements * sizeof(int));
	hipMalloc(&d_b, num_elements * sizeof(int));
	hipMalloc(&d_c, num_elements * sizeof(int));
	CheckCudaError("malloc_device_error");


	/*Copiamos los elementos del host ya llenados anteriormente (llenamos memoria,
		copiando las matrizes del host hacia la tarjeta gráfica (device).*/
	hipMemcpy(d_a, h_a, num_elements * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, num_elements * sizeof(int), hipMemcpyHostToDevice);
	CheckCudaError("memcpy_error");


	free(h_b); 
	CheckCudaError("Free_host_error");


	//Establecemos el num de threas y blocks que utilizaremos
	dim3 gridDim (NumBlocks, NumBlocks);
	dim3 blockDim (BLOCK_SIZE, BLOCK_SIZE);
	//LLamamos la función.
	productMatrix <<< gridDim, blockDim >>> (d_a, d_b, d_c);
	CheckCudaError("Calling_device_function_error");


	/*Esperamos a que todos los threads hayan hecho su trabajo (multiplicar las matrizes)
		antes de copy back.*/
	cudaThreadSyncronize();
	CheckCudaError("Syncronize_threads_error");


	//Una vez sincronizados los volvemos a copiar hacia el host.
	hipMemcpy(h_a, d_c, num_elements * sizeof(int), hipMemcpyDeviceToHost);
	CheckCudaError("mempcy_host_error");


	//Imprimimos por pantalla
	for (int i = 0; i < num_elements; ++i) cout << h_a[i];


	//Aliberamos memoria en el device
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	//Aliveramos meomria en host.
	free(h_a);

	CheckCudaError("free_device_error");

}


void CheckCudaError(string &e)
{	
	//Obtenemos el ultimo error.
	hipError_t err = hipGetLastError();
	//Si hay error imprime el error por pantalla
	if(hipSuccess != err){
		cout << e << endl;
	}
}
